#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>

// CUDA runtime
#include "helper.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>


/*
naive 版每个 thread 都在做 global_mem -------> reg 的超远距离（473 cycle 延迟）搬运，第二版本使用 __shared__ 声明静态 share_memory ， preload 16x16 小块的正方形，多个 thread 共用，少 load gmem。

https://zhuanlan.zhihu.com/p/342103911  

将矩阵分块，读到share_mem的块会被访问多次（减少global_mem的次数）
*/
// a = mxk, b = kxn
template <int BLOCK>
__global__ void sgemm(int m, int n, int k, float *a, int lda, float *b, int ldb,
                      float *c, int ldc) {
  // blockIdx control subpanel matrix

  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int bx = blockIdx.x;
  const int by = blockIdx.y;

  float *begin_a = a + by * BLOCK * k;
  float *begin_b = b + bx * BLOCK;
  float *end_a = begin_a + k;

  float sum = 0.f;
  for (float *a_ptr = begin_a, *b_ptr = begin_b; a_ptr < end_a;
       a_ptr += BLOCK, b_ptr += BLOCK * n) {
    __shared__ float ashare[BLOCK][BLOCK];
    __shared__ float bshare[BLOCK][BLOCK];

    ashare[ty][tx] = a_ptr[ty * k + tx];
    bshare[ty][tx] = b_ptr[ty * n + tx];
    __syncthreads();

// #pragma unroll
    for (int kk = 0; kk < BLOCK; ++kk) {
      sum += ashare[ty][kk] * bshare[kk][tx];
    }
    __syncthreads();
  }

  c[(BLOCK * by + ty) * n + BLOCK * bx + tx] = sum;
}

void MY_MMult(hipblasHandle_t handle, int m, int n, int k, float *d_A, int lda,
              float *d_B, int ldb, float *d_C, int ldc) {

  constexpr int BLOCK = 16;
  dim3 block(BLOCK, BLOCK);
  dim3 grid((m + BLOCK - 1) / BLOCK, (n + BLOCK - 1) / BLOCK);

  sgemm<BLOCK><<<grid, block>>>(m, n, k, d_A, lda, d_B, ldb, d_C, ldc);
}
