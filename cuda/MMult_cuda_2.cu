#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>

// CUDA runtime
#include "helper.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

/**
 * naive 实现，tilling without share mem 
 */
template <int BLOCK>
__global__ void sgemm(int m, int n, int k, float *a, int lda, float *b, int ldb,
                      float *c, int ldc) {
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  int by = blockIdx.y;
  float *begin_a = a+
}

void MY_MMult(hipblasHandle_t handle, int m, int n, int k, float *d_A, int lda,
              float *d_B, int ldb, float *d_C, int ldc) {

  constexpr int BLOCK = 16;
  // subm, subn, subk
  dim3 block(BLOCK, BLOCK);
  dim3 grid((m + BLOCK - 1) / BLOCK, (n + BLOCK - 1) / BLOCK);

  sgemm<BLOCK><<<grid, block>>>(m, n, k, d_A, lda, d_B, ldb, d_C, ldc);
}
